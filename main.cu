#include "hip/hip_runtime.h"
%%cu
#include <fstream>
#include <iostream>
#include <cmath>
#include <vector>
using namespace std;

#define sqr(x) ((x) * (x))

static constexpr double EPS = 0.01;
static constexpr double F_EPS = 1e-6;
static const int INF = 1e9 + 7;


namespace Hartman3 {
    double a[4][3];
    double p[4][3];

    double c[4];

    int GetDimensions() {
        return 3;
    }

    void InitializeBorders(int** borders) {
        for (int i = 0; i < 3; ++i) {
            borders[i][0] = 0;
            borders[i][1] = 1;
        }
    }

    void AdditionalInitialize() {
        double A[4][3] = {
          { 3, 10, 30},
          { 0.1, 10, 35},
          { 3, 10, 30},
          { 0.1, 10, 35}
        };
        for (int i = 0; i < 4; ++i) {
          hipMemcpyToSymbol(HIP_SYMBOL(a[i]), A[i], sizeof(double) * 3);
        }
        double P[4][3] = {
          { 0.3689, 0.1170, 0.2673},
          { 0.4699, 0.4387, 0.7470},
          { 0.1091, 0.8732, 0.5547},
          { 0.03815, 0.5743, 0.8828}
        };
        for (int i = 0; i < 4; ++i) {
          hipMemcpyToSymbol(HIP_SYMBOL(p[i]), P[i], sizeof(double) * 3);
        }
        const double C[4] = {1.0, 1.2, 3.0, 3.2};
        hipMemcpyToSymbol(HIP_SYMBOL(c), C, sizeof(double) * 4);
    }

    __device__
    double func(double* x) {
        double y = 0.0;
        for (int i = 0; i < 4; i++) {
            double e = 0.0;
            for (int j = 0; j < 3; j++)
                e += a[i][j] * sqr(x[j] - p[i][j]);
            printf("c[i] = %lf\n", c[i]);
            y += c[i] * exp(-e);
        }
        return -y;
    }
} 

namespace Ackley3 {
    // bounds = [-32, 32], [-32, 32], min = 82.4617 at (-32, -32)

    int GetDimensions() {
        return 2;
    }

    void InitializeBorders(int** borders) {
        int bordersHost[2][2] = {{-32000, 32000}, {-32000, 32000}};

        for (int i = 0; i < 2; ++i) {
            hipMemcpy(borders[i], bordersHost[i], sizeof(int) * 2, hipMemcpyHostToDevice);
        }
    }

    __device__
    double func(double* x) {
        return 200 * exp(-0.02 * sqrt(sqr(x[0]) + sqr(x[1]))) + 5 * exp(cos(3 * x[0]) + sin(3 * x[1]));
    }
}

namespace StyblinskiTang {
    int GetDimensions() {
        return 2;
    }

    void InitializeBorders(int** borders) {
        int bordersHost[2][2] = {{-5, 5}, {-5, 5}};

        for (int i = 0; i < 2; ++i) {
            hipMemcpy(borders[i], bordersHost[i], sizeof(int) * 2, hipMemcpyHostToDevice);
        }
    }

    __device__
    double func(double* x) {
        return 0.5 * (pow(x[0], 4) - 16 * sqr(x[0]) + 5 * x[0] + pow(x[1], 4) - 16 * sqr(x[1]) + 5 * x[1]);
    }
}

namespace Beale {
    int GetDimensions() {
        return 2;
    }

    void InitializeBorders(int** borders) {
        int bordersHost[2][2] = {{-5, 5}, {-5, 5}};

        for (int i = 0; i < 2; ++i) {
            hipMemcpy(borders[i], bordersHost[i], sizeof(int) * 2, hipMemcpyHostToDevice);
        }
    }

    __device__
    double func(double* x) {
        return sqr(1.5 - x[0] + x[0] * x[1]) + sqr(2.25 - x[0] + x[0] * sqr(x[1])) + sqr(2.625 - x[0] + x[0] * (x[1] * x[1] * x[1]));
    }
}

namespace EggHolder {
    int GetDimensions() {
        return 2;
    }

    void InitializeBorders(int** borders) {
        int bordersHost[2][2] = {{-512, 512}, {-512, 512}};

        for (int i = 0; i < 2; ++i) {
            hipMemcpy(borders[i], bordersHost[i], sizeof(int) * 2, hipMemcpyHostToDevice);
        }
    }

    __device__
    double func(double* x) {
        return -(x[1] + 47) * sin(sqrt(abs(x[1] + x[0] / 2.0 + 47))) - x[0] * sin(sqrt(abs(x[0] - x[1] - 47)));
    }
}

namespace Rosenbrock {
    int GetDimensions() {
        return 3;
    }

    void InitializeBorders(int** borders) {
        int bordersHost[3][2] = {{-30, 30}, {-30, 30}, {-30, 30}};

        for (int i = 0; i < 3; ++i) {
            hipMemcpy(borders[i], bordersHost[i], sizeof(int) * 3, hipMemcpyHostToDevice);
        }
    }

    __device__
    double func(double* x) {
        return sqr(1 - x[0]) + 100 * sqr(x[1] - sqr(x[0])) + sqr(1 - x[1]) + 100 * sqr(x[2] - sqr(x[1]));
    }
}

__device__
double atomicMin(double* address, double val) {
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(min(val,
                               __longlong_as_double(assumed))));
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__
double f(double* x) {
    return Beale::func(x);
}

__device__
double calcL(double* x, double* y, double step) {
    return fabs(f(x) - f(y)) / step;
}

__device__
double calcK(double diameter) {
    return exp(diameter);
}

__device__
bool checkStop(int k, int lastModified, double f_rec, double f_pred, double L_rec, double* threadWidth, double dimensions, double error, double diameter, double* globalResult) {
    if (k == 2) {
        if (diameter * L_rec < 2 * (error + f_rec - *globalResult)) {
            return true;
        }
        return false;
    }
    if (k - lastModified > 3) {
        return true;
    }

    return diameter * L_rec < 2 * (error + f_rec - *globalResult) || fabs(f_pred - f_rec) < error || fabs(f_pred - f_rec) / f_pred < EPS;
}

__device__ double calcLStart(int dimensions, double* x_center, double* x_next, double* x_0, double* threadWidth, double step) {
    if (dimensions == 2) {
        double L_rec = 0;
        for (int i = 0; i < dimensions; ++i) {
            x_next[i] = x_0[i];
        }
        L_rec = calcL(x_center, x_next, step / 2);
        x_next[0] += threadWidth[0];
        L_rec = max(L_rec, calcL(x_center, x_next, step / 2));
        x_next[1] += threadWidth[1];
        L_rec = max(L_rec, calcL(x_center, x_next, step / 2));
        x_next[0] = x_0[0];
        L_rec = max(L_rec, calcL(x_center, x_next, step / 2));
        return L_rec;
    }
    double L_rec = 0;
    int diff[8][3] = {{0, 0, 1}, {0, 1, 0}, {0, 1, 1}, {1, 0, 0}, {1, 0, 1}, {1, 1, 0}, {1, 1, 1}};
    for (int i = 0; i < 8; ++i) {
        int mult = 1;
        for (int j = 0; j < 2; ++j) {
            mult *= -1;
            for (int k = 0; k < dimensions; ++k) {
                x_next[k] = x_0[k] + mult * diff[i][k];
            }
            L_rec = max(L_rec, calcL(x_center, x_next, step / 2));
        }
    }
    return L_rec;
}

__global__
void apply(int** borders, int* blockCount, int blockWidth, int blockSize, int dimensions, double* threadWidth,
           double error, double* globalResult, unsigned int* steps, double diameter, double* results) {
    int idxBlock = blockIdx.x;
    int idxThread = threadIdx.x;
    int index;
    double x_center[3];
    double x_0[3];
    double x_cur[3];
    double x_next[3];
    for (int i = 0; i < dimensions; ++i) {
        index = (idxBlock % blockCount[i]) * blockWidth + idxThread % blockWidth;
        x_0[i] = index * threadWidth[i] + borders[i][0];
        x_center[i] = x_0[i] + (threadWidth[i] / 2.0);
        idxThread /= blockWidth;
        idxBlock /= blockCount[i];
    }
    // calculate f() in the center
    double f_rec = f(x_center);
    atomicMin(globalResult, f_rec);
    double L_rec = calcLStart(dimensions, x_center, x_next, x_0, threadWidth, diameter);
    int k = 2;
    double L_cur;
    double f_cur;
    double f_pred = INF;
    int lastModified = 2;
    int stepsCur = 0;
    while (!checkStop(k, lastModified, f_rec, f_pred, L_rec, threadWidth, dimensions, error, diameter, globalResult)) {
        for (int j = 0; j < k; ++j) {
            for (int i = 0; i < dimensions; ++i) {
                x_cur[i] = x_0[i] + (threadWidth[i] / k) * j;
                x_next[i] = x_cur[i];
            }
            f_cur = f(x_cur);
            if (f_cur < f_rec - F_EPS) {
                lastModified = k;
                f_pred = f_rec;
                f_rec = f_cur;
            }
            for (int i = 0; i < dimensions; ++i) {
                x_next[i] += threadWidth[i] / k;
                L_cur = calcL(x_cur, x_next, threadWidth[i] / k);
                if (L_cur > L_rec) {
                    L_rec = L_cur;
                }
            }
        }
        stepsCur += k;
        k++;
    }
    results[blockIdx.x * blockSize + threadIdx.x] = f_rec;
    steps[blockIdx.x * blockSize + threadIdx.x] = k - 1;
    atomicMin(globalResult, f_rec);
}


int runTest(int* blockCountH, double error, int blockSize, double* time, double* averageSteps) {
    int dimensions = Beale::GetDimensions();
    int** borders;
    hipMallocManaged(&borders, dimensions * sizeof(int*));
    for (int i = 0; i < dimensions; ++i) {
        hipMallocManaged(&borders[i], sizeof(int) * 2);
    }

    Beale::InitializeBorders(borders);

    int* blockCount;
    hipMallocManaged(&blockCount, sizeof(int) * dimensions);
    int blocksNumAll = 1;
    for (int i = 0; i < dimensions; ++i) {
         blocksNumAll *= blockCountH[i];
    }
    hipMemcpy(blockCount, blockCountH, sizeof(int) * dimensions, hipMemcpyHostToDevice);

    int blockWidth = sqrt(blockSize);
    if (dimensions == 3) {
        blockSize = 64;
        blockWidth = 4;
    }
    double* threadWidth;
    double* globalResult;
    double* results;
    unsigned int* steps;

    hipMallocManaged(&threadWidth, sizeof(double) * dimensions);
    hipMallocManaged(&globalResult, sizeof(double));
    hipMallocManaged(&steps, sizeof(unsigned int) * blockSize * blocksNumAll);
    hipMallocManaged(&results, sizeof(double) * blockSize * blocksNumAll);
    double* threadWidthHost = (double*)malloc(sizeof(double) * dimensions);
    double diameter = 0;
    for (int i = 0; i < dimensions; ++i) {
        threadWidthHost[i] = (borders[i][1] - borders[i][0]) / (double)(blockCount[i] * blockWidth);
        diameter += sqr(threadWidthHost[i]);
    }
    diameter = sqrt(diameter);

    hipMemcpy(threadWidth, threadWidthHost, sizeof(double) * dimensions, hipMemcpyHostToDevice);

    double globalResultHost[1];
    globalResultHost[0] = INF;
    hipMemcpy(globalResult, globalResultHost, sizeof(double), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    apply<<<blocksNumAll, blockSize>>>(borders, blockCount, blockWidth, blockSize, dimensions, threadWidth, error, globalResult, steps, diameter, results);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    double ans[2];
    hipMemcpy(ans, globalResult, sizeof(double), hipMemcpyDeviceToHost);
    unsigned int stepCount[blocksNumAll * blockSize];
    hipMemcpy(stepCount, steps, sizeof(unsigned int) * blocksNumAll * blockSize, hipMemcpyDeviceToHost);
    double resultsGlobal[blocksNumAll * blockSize];
    hipMemcpy(resultsGlobal, results, sizeof(double) * blocksNumAll * blockSize, hipMemcpyDeviceToHost);
    double record = INF;
    for (int i = 0; i < blocksNumAll * blockSize; ++i) {
        record = min(record, resultsGlobal[i]);
    }
    int allSteps = 0;
    for (int i = 0; i < blocksNumAll * blockSize; ++i) {
        allSteps += stepCount[i];
    }
    cout << "Global min value is " << record << ", steps = " << allSteps 
         << ", avgSteps = " << (double)allSteps / (blocksNumAll * blockSize) << ", threadsCount = " << (blocksNumAll * blockSize) << endl;
    float timeSpent;
    hipEventElapsedTime(&timeSpent, start, stop);
    cout << timeSpent << " " << allSteps << endl;
    *time = timeSpent;
    *averageSteps = (double)allSteps / (blocksNumAll * blockSize);

    for (int i = 0; i < dimensions; ++i) {
        hipFree(borders[i]);
    }
    free(threadWidthHost);
    hipFree(borders);
    hipFree(blockCount);
    hipFree(threadWidth);

    return allSteps;
}


int main() {
    double error = 0.001;
    int blockCount[3];
    int BLOCKS = 5;
    int blockSize = 64;
    double timeSpent;
    double avgSteps;
    ofstream fout("beale_threads_16_new");
    for (int i = 2; i < 3; ++i) {
        blockCount[0] = blockCount[1] = blockCount[2] = i;
        int threadsNumAll = i * i * blockSize;
        int iterCount = runTest(blockCount, error, blockSize, &timeSpent, &avgSteps);
        fout << i << " " << iterCount << " " << timeSpent << " " << avgSteps << " " << threadsNumAll << endl;
    }
    fout.close();
    return 0;
}
